#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <valhalla.hpp>

namespace vll {

    template<typename T>
    void reduce(const int num, bool gen = true, int iter = 0)
    {
        static thrust::device_vector<T> d_vec;

        if (gen) {
            thrust::host_vector<T> h_vec(num);
            std::generate(h_vec.begin(), h_vec.end(), rand);
            d_vec = h_vec;
            hipDeviceSynchronize();
        }

        for (int i = 0; i < iter; i++) {
            T x = thrust::reduce(d_vec.begin(), d_vec.end());
        }

        hipDeviceSynchronize();
    }

    template<typename T>
    void generate(const int num)
    {
        try {
            reduce<T>(num, true, 0);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }

    template<typename T>
    void run(const int iter)
    {
        try {
            reduce<T>(-1, false, iter);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }
}

int main(int argc, const char **args)
{
    vll::launch(argc, args);
}
