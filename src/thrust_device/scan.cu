#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <valhalla.hpp>

namespace vll {

    template<typename T>
    void scan(const int num, bool gen = true, int iter = 0)
    {
        static thrust::device_vector<T> d_in;
        static thrust::device_vector<T> d_out;

        if (gen) {
            thrust::host_vector<T> h_in(num);
            std::generate(h_in.begin(), h_in.end(), rand);
            d_in = h_in;
            d_out = thrust::device_vector<T>(num);
            hipDeviceSynchronize();
        }

        for (int i = 0; i < iter; i++) {
            thrust::inclusive_scan(d_in.begin(), d_in.end(), d_out.begin());
        }

        hipDeviceSynchronize();
    }

    template<typename T>
    void generate(const int num)
    {
        try {
            scan<T>(num, true, 0);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }

    template<typename T>
    void run(const int iter)
    {
        try {
            scan<T>(-1, false, iter);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }
}

int main(int argc, const char **args)
{
    vll::launch(argc, args);
}
