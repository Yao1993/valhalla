#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <valhalla_improved.hpp>

namespace vll {

    template <typename T>
    class SortWrapper : public WrapperBase
    {
        thrust::device_vector<T> d_data0;
        thrust::device_vector<T> d_data;

	public:
        void generate(const int num) override
        {
            thrust::host_vector<T> h_data(num);
			generate_raw_data(&h_data[0], num);
            d_data0 = h_data; 
            hipDeviceSynchronize();
        }

        void reset() override
        {         
            d_data = d_data0;
            hipDeviceSynchronize();
        }

        void run() override
        {
            thrust::sort(d_data.begin(), d_data.end());
            hipDeviceSynchronize();            
        }
    };


	WrapperBase* sort_warpper;

    template<typename T>
    void generate(const int num)
    {
		sort_warpper = new SortWrapper<T>();
        try {
           sort_warpper->generate(num);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }

    template<typename T>
    void reset()
    {
        try {
            sort_warpper->reset();
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }

    template<typename T>
    void run()
    {
        try {
            sort_warpper->run();
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }
}

int main(int argc, const char **args)
{
    vll::launch(argc, args);
}
