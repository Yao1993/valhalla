#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <valhalla.hpp>

namespace vll {

    template<typename T>
    void inner_product(const int num, bool gen = true, int iter = 0)
    {
        static thrust::device_vector<T> d_A;
        static thrust::device_vector<T> d_B;
        

        if (gen) {
            thrust::host_vector<T> h_A(num);
            thrust::host_vector<T> h_B(num);            
            std::generate(h_A.begin(), h_A.end(), rand);
            std::generate(h_B.begin(), h_B.end(), rand);            
            d_A = h_A;
            d_B = h_B;
            hipDeviceSynchronize();
        }

        for (int i = 0; i < iter; i++) {
            T x = thrust::inner_product(d_A.begin(), d_A.end(), d_B.begin(), T());
        }

        hipDeviceSynchronize();
    }

    template<typename T>
    void generate(const int num)
    {
        try {
            inner_product<T>(num, true, 0);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }

    template<typename T>
    void run(const int iter)
    {
        try {
            inner_product<T>(-1, false, iter);
        } catch(const thrust::system_error &ex) {
            std::cout << ex.what() << std::endl;
        }
    }
}

int main(int argc, const char **args)
{
    vll::launch(argc, args);
}
