#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <valhalla.hpp>

namespace vll {

    template<typename T>
    struct saxpy_functor : public thrust::binary_function<T, T, T>
    {
        const T a;

        saxpy_functor(T _a) : a(_a) {}

            __host__ __device__
            T operator()(const T& x, const T& y) const {
                return a * x + y;
            }
    };

    template<typename T>
    void saxpy(const int num, bool gen = true, int iter = 0)
    {
        static thrust::device_vector<T> x;
        static thrust::device_vector<T> y;
        static thrust::device_vector<T> res;
        static T a = 3.5;

        if (gen) {
            thrust::host_vector<T> h_x(num);
            std::generate(h_x.begin(), h_x.end(), rand);

            thrust::host_vector<T> h_y(num);
            std::generate(h_y.begin(), h_y.end(), rand);

            x = h_x;
            y = h_y;
            res = thrust::device_vector<T>(num);
            hipDeviceSynchronize();
        }

        for (int i = 0; i < iter; i++) {
            thrust::transform(x.begin(), x.end(),
                              y.begin(), res.begin(),
                              saxpy_functor<T>(a));
        }
        hipDeviceSynchronize();
    }

    template<typename T>
    void generate(const int num)
    {
        try {
            saxpy<T>(num, true, 0);
        } catch(thrust::system::system_error &err) {
            std::cout <<  err.what() << std::endl;
        }
    }

    template<typename T>
    void run(const int iter)
    {
        try {
            saxpy<T>(-1, false, iter);
        } catch(thrust::system::system_error &err) {
            std::cout <<  err.what() << std::endl;
        }
    }
}

int main(int argc, const char **args)
{
    vll::launch(argc, args);
}
